#include "hip/hip_runtime.h"

#include "user_nn_matrix_cuda.h"
#include "user_nn_matrix.h"

//__global__ ��������
//__device__ �豸����
//transpose_naive << < 1, matrix_h >> >(sub_matrix, src_matrix, matrix_w, matrix_h);
//extern "C" template <int BLOCK_SIZE>
//extern "C"
__global__ void transposeNaive_array(float *odata, float* idata, int width, int height)
{
	//threadId�̵߳����� blockDim�߳̿��ά�� gridDim�̸߳��ά��
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;//grid���ֳ�1ά��block����Ϊ1ά ��ȡ������
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;//
	unsigned int index_in = xIndex + width * yIndex;//�����߳����� ���Ե�ַ
	unsigned int index_out = yIndex + height * xIndex;

	unsigned int width_index = 0;
	for (width_index = 0; width_index < height; width_index++){
		odata[index_out + width_index] = idata[index_in + width_index * width];//����һ��������һ�е�λ��
	}
	/*
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	if (xIndex < width && yIndex < height)
	{
		unsigned int index_in = xIndex + width * yIndex;
		unsigned int index_out = yIndex + height * xIndex;
		odata[index_out] = idata[index_in];
	}
	*/
}

__global__ void transposeNaive_block(float *odata, float *idata, int width, int height)
{
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

	int index_in = xIndex + width * yIndex;
	int index_out = yIndex + height * xIndex;

	for (int i = 0; i<TILE_DIM; i += BLOCK_ROWS)
	{
		odata[index_out + i] = idata[index_in + i*width];
	}
}

//����ת��
extern "C"
void user_nn_matrix_transpose_cuda(user_nn_matrix *src_matrix){
	int src_count = src_matrix->width * src_matrix->height;
	float *cuda_input_matrix = NULL;
	float *cuda_output_matrix = NULL;

	hipMalloc((void **)&cuda_input_matrix, src_count * sizeof(float));//�������ݿռ�
	hipMalloc((void **)&cuda_output_matrix, src_count * sizeof(float));//�������ݿռ�

	hipMemcpy(cuda_input_matrix, src_matrix->data, src_count * sizeof(float), hipMemcpyHostToDevice);//��������
	//<<<grid, threads>>> grid��ʾ�̸߳��ж��ٶ��ٸ��߳̿飬threads��ʾһ���߳̿����ж��ٸ��߳�
	if ((src_matrix->width%TILE_DIM == 0) && (src_matrix->height%TILE_DIM == 0) && (src_matrix->width == src_matrix->height)){
		dim3 grid(src_matrix->width / TILE_DIM, src_matrix->height / TILE_DIM), threads(TILE_DIM, BLOCK_ROWS);//��̬��������Ҫ��դ����߳�
		transposeNaive_block << <grid, threads >> >(cuda_output_matrix, cuda_input_matrix, src_matrix->width, src_matrix->height);//���о���ת��
	}
	else{
		transposeNaive_array << <1, ((src_matrix->width > src_matrix->height) ? src_matrix->width : src_matrix->height) >> >(cuda_output_matrix, cuda_input_matrix, src_matrix->width, src_matrix->height);//���о���ת��
	}
	hipMemcpy(src_matrix->data, cuda_output_matrix, src_count * sizeof(float), hipMemcpyDeviceToHost);//��������
	//��������λ��
	src_matrix->width  = src_matrix->width ^ src_matrix->height;
	src_matrix->height = src_matrix->width ^ src_matrix->height;
	src_matrix->width  = src_matrix->width ^ src_matrix->height;

	hipFree(cuda_input_matrix);
	hipFree(cuda_output_matrix);
}
#ifdef WIN64
user_nn_matrix * user_nn_matrix_mult_matrix_cuda(user_nn_matrix *src_matrix, user_nn_matrix *sub_matrix){
	hipblasHandle_t cuda_handle;
	hipblasStatus_t status;
	hipError_t error;

	user_nn_matrix *result = NULL;//�������
	int src_count = src_matrix->width * src_matrix->height;
	int sub_count = sub_matrix->width * sub_matrix->height;
	float *src_matrix_cuda = NULL, *sub_matrix_cuda = NULL, *result_cuda = NULL;
	float alpha = 1.0, beta = 0.0;

	if (src_matrix->width != sub_matrix->height){//����˻�ֻ�е���һ�����������=�ڶ��������������������
		return NULL;
	}
	status = hipblasCreate(&cuda_handle);
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("status error");
	}

	result = user_nn_matrix_create(sub_matrix->width, src_matrix->height);//�����µľ���

	error = hipMalloc((void **)&src_matrix_cuda, src_count * sizeof(float));//�������ݿռ�
	error = hipMalloc((void **)&sub_matrix_cuda, sub_count * sizeof(float));//�������ݿռ�
	error = hipMalloc((void **)&result_cuda, result->width * result->height * sizeof(float));//���䱣�����ľ���

	error = hipMemcpy(src_matrix_cuda, src_matrix->data, src_count * sizeof(float), hipMemcpyHostToDevice);//��������
	error = hipMemcpy(sub_matrix_cuda, sub_matrix->data, sub_count * sizeof(float), hipMemcpyHostToDevice);//��������

	//user_nn_matrix_memset(result,1.2f);
	//hipMemcpy(result_cuda, result->data, result->width * result->height * sizeof(float), hipMemcpyHostToDevice);//��������

	//��ʽ��C = alpha*op(A)xop(B)+beta*C
	//�����A�Ƿ�ת�á�B�Ƿ�ת�á�����A��������A������A����������B��alpha��Aָ�롢lda��B��ldb��beta��C��ldc
	status = hipblasSgemm(cuda_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, result->height, result->width, src_matrix->width, &alpha, src_matrix_cuda, src_matrix->width, sub_matrix_cuda, sub_matrix->width, &beta, result_cuda, result->height);
	error = hipDeviceSynchronize();

	error = hipMemcpy(result->data, result_cuda, result->width * result->height * sizeof(float), hipMemcpyDeviceToHost);//��������


	hipFree(src_matrix_cuda);
	hipFree(sub_matrix_cuda);
	hipFree(result_cuda);
	hipblasDestroy(cuda_handle);

	return result;
}
#endif